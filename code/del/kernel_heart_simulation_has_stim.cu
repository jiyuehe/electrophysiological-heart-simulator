
#include <hip/hip_runtime.h>
__global__ void simulation_kernel(double* u_next, double* h_next, const double* u_current, const double* h_current, const int t_id, const double dt, const double delta,
	const int* indices, const double* parts, const int N, const int L, const int* voxel_id, const int* time_id, const double* non_zero_value, const int n_voxel) {
    // note: the first 2 arguments are input and output arguments, the rest arguments with "const" are input arguments only
    
	int v_id = threadIdx.x + (blockIdx.x * blockDim.x); // blockDim.x is the built-in variable for threads per block

    if (v_id > n_voxel - 1) {
		return;
	}

    // NOTE: a^3 should write as a*a*a, or will have compile error
	double diffusion_term = 1.0 / (4.0 * delta * delta) *
		(parts[v_id * 21 + 0] * (u_current[indices[v_id * 18 + 0]] - u_current[v_id]) + parts[v_id * 21 + 1] * (u_current[indices[v_id * 18 + 1]] - u_current[v_id]) +
			parts[v_id * 21 + 2] * (u_current[indices[v_id * 18 + 2]] - u_current[v_id]) + parts[v_id * 21 + 3] * (u_current[indices[v_id * 18 + 3]] - u_current[v_id]) +
			parts[v_id * 21 + 4] * (u_current[indices[v_id * 18 + 4]] - u_current[v_id]) + parts[v_id * 21 + 5] * (u_current[indices[v_id * 18 + 5]] - u_current[v_id]) +
			parts[v_id * 21 + 6] * (u_current[indices[v_id * 18 + 0]] - u_current[indices[v_id * 18 + 1]]) +
			parts[v_id * 21 + 7] * (u_current[indices[v_id * 18 + 2]] - u_current[indices[v_id * 18 + 3]]) +
			parts[v_id * 21 + 8] * (u_current[indices[v_id * 18 + 4]] - u_current[indices[v_id * 18 + 5]]) +
			parts[v_id * 21 + 9] * (u_current[indices[v_id * 18 + 6]] - u_current[indices[v_id * 18 + 8]]) + parts[v_id * 21 + 10] * (u_current[indices[v_id * 18 + 9]] - u_current[indices[v_id * 18 + 7]]) +
			parts[v_id * 21 + 11] * (u_current[indices[v_id * 18 + 14]] - u_current[indices[v_id * 18 + 16]]) + parts[v_id * 21 + 12] * (u_current[indices[v_id * 18 + 17]] - u_current[indices[v_id * 18 + 15]]) +
			parts[v_id * 21 + 13] * (u_current[indices[v_id * 18 + 10]] - u_current[indices[v_id * 18 + 12]]) + parts[v_id * 21 + 14] * (u_current[indices[v_id * 18 + 13]] - u_current[indices[v_id * 18 + 11]]));
	diffusion_term = parts[v_id * 21 + 20] * diffusion_term;

	// J_stim
	int id = -1;
	for (int i = 0; i < L; i++) {
		if (v_id == voxel_id[i]) { // is pacing voxel
			id = i;
		}
	}
	double J_stim = 0.0;
	if (id != -1) {
		J_stim = non_zero_value[id];
	}

	// update u
	u_next[v_id] = ((h_current[v_id] * u_current[v_id] * u_current[v_id] * (1 - u_current[v_id]) / parts[v_id * 21 + 17]) + (-u_current[v_id] / parts[v_id * 21 + 18]) + J_stim + diffusion_term) * dt + u_current[v_id];

	// update h
	if (u_current[v_id] < parts[v_id * 21 + 19]) {
		h_next[v_id] = ((1 - h_current[v_id]) / parts[v_id * 21 + 15]) * dt + h_current[v_id];
	}
	if (u_current[v_id] >= parts[v_id * 21 + 19]) {
		h_next[v_id] = (-h_current[v_id] / parts[v_id * 21 + 16]) * dt + h_current[v_id];
	}

	return;
}
