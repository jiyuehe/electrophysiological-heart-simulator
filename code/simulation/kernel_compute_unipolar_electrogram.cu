
#include <hip/hip_runtime.h>
__global__ void egm_kernel(double* egm_part, const double* P, const int n_node, const int n_parts, const double* dvdx, const double* dvdy, const double* dvdz) {
    // note: the first argument is input and output argument, the argument with "const" is input argument only
    
	int n_id = threadIdx.x + (blockIdx.x * blockDim.x); // blockDim.x is the built-in variable for threads per block

    if (n_id > n_node - 1) {
		return;
	}

    // NOTE: a^3 should write as a*a*a, or will have compile error
    // matlab index starts at 1, c index starts at 0
    egm_part[n_id] = P[n_id*n_parts+1-1] / (P[n_id*n_parts+11-1]*P[n_id*n_parts+11-1]*P[n_id*n_parts+11-1]) *
        ( (P[n_id*n_parts+2-1] * dvdx[n_id] + P[n_id*n_parts+3-1] * dvdy[n_id] + P[n_id*n_parts+4-1] * dvdz[n_id]) * P[n_id*n_parts+12-1] +
        (P[n_id*n_parts+5-1] * dvdx[n_id] + P[n_id*n_parts+6-1] * dvdy[n_id] + P[n_id*n_parts+7-1] * dvdz[n_id]) * P[n_id*n_parts+13-1] +
        (P[n_id*n_parts+8-1] * dvdx[n_id] + P[n_id*n_parts+9-1] * dvdy[n_id] + P[n_id*n_parts+10-1] * dvdz[n_id]) * P[n_id*n_parts+14-1] );

	return;
}
